/*******************************************************************************
                        cuda functions for gpu processing
*******************************************************************************/

#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/videoio.hpp>
#include <opencv2/imgproc.hpp>

#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>

#include "video_param.h"

/******************************************************************************/

// __device__ void cufftReal_convert(cv::Mat * d_mat, hipfftReal * d_raw) {
//
//   d_raw[blockIdx.x] = (hipfftReal) d_mat->at<double>(blockIdx.x);
// }
  

bool fft_batched(hipfftReal * d_raw, video_param_t video_param, hipfftComplex * d_ftd) {

  // create plan for performing fft
  hipfftHandle plan;
  size_t batch = video_param.height * video_param.width;
  size_t n_points = video_param.n_frames;
  if (hipfftPlan1d(&plan, n_points, HIPFFT_R2C, batch) != HIPFFT_SUCCESS) {
    printf("Failed to create 1D plan\n");
    return -1;
  }

  // allocate return data
  hipMalloc((void**) &d_ftd, sizeof(hipfftComplex)*n_points * batch);
  if (hipGetLastError() != hipSuccess) {
    printf("Failed to allocate memory space for transformed data.\n");
    return -1;
  }

  // perform fft
  if (hipfftExecR2C(plan, d_raw, d_ftd) != HIPFFT_SUCCESS) {
    printf("Failed to perform fft.\n");
    return -1;
  }

  hipfftDestroy(plan);
  hipFree(d_raw);

  return 0;
}

/******************************************************************************/
