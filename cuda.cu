/*******************************************************************************
cuda functions for backend of fourier transforms
*******************************************************************************/

__device__ void cufftReal_convert(cv::Mat * d_mat, hipfftReal * d_raw) {

  d_raw[blockIdx.x] = (hipfftReal) d_mat->at(blockIdx.x);
}
